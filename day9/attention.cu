#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>

#include "C:\Users\radio\Documents\GitHub\100-days-CUDA\day9\include\matmul.cuh"
#include "C:\Users\radio\Documents\GitHub\100-days-CUDA\day9\include\transpose.cuh"
#include "C:\Users\radio\Documents\GitHub\100-days-CUDA\day9\include\softmax.cuh"

#define BATCH 3
#define HEADS 8
#define SEQ_LEN 4096
#define D_K 32

int main() {
    int B = BATCH, H = HEADS, S = SEQ_LEN, D = D_K;
    int total_heads = B * H;

    int qkv_size = total_heads * S * D;
    int attn_scores_size = total_heads * S * S;
    int output_size = total_heads * S * D;

    // CUDA bellekler
    float *Q, *K, *V, *K_T, *attn_scores, *softmax_output, *final_output;

    hipMalloc(&Q, qkv_size * sizeof(float));
    hipMalloc(&K, qkv_size * sizeof(float));
    hipMalloc(&V, qkv_size * sizeof(float));
    hipMalloc(&K_T, qkv_size * sizeof(float)); // transposed K
    hipMalloc(&attn_scores, attn_scores_size * sizeof(float));
    hipMalloc(&softmax_output, attn_scores_size * sizeof(float));
    hipMalloc(&final_output, output_size * sizeof(float));

    // Doldur Q, K, V
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng, 1234ULL);
    hiprandGenerateUniform(prng, Q, qkv_size);
    hiprandGenerateUniform(prng, K, qkv_size);
    hiprandGenerateUniform(prng, V, qkv_size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float t_transpose = 0, t_qk_matmul = 0, t_softmax = 0, t_v_matmul = 0;

    hipEventRecord(start);
    for (int i = 0; i < total_heads; i++) {
        launch_transpose(
            K + i * S * D,
            K_T + i * D * S,
            D, S
        );
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&t_transpose, start, stop);
    std::cout << "Transpose time: " << t_transpose << " ms" << std::endl;

    hipEventRecord(start);
    for (int i = 0; i < total_heads; i++) {
        launch_matmul(
            Q + i * S * D,
            K_T + i * D * S,
            attn_scores + i * S * S,
            S, S, D
        );
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&t_qk_matmul, start, stop);
    std::cout << "Q × K^T time: " << t_qk_matmul << " ms" << std::endl;

    hipEventRecord(start);
    launch_softmax(attn_scores, softmax_output, total_heads * S, S);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&t_softmax, start, stop);
    std::cout << "Softmax time: " << t_softmax << " ms" << std::endl;

    hipEventRecord(start);
    for (int i = 0; i < total_heads; i++) {
        launch_matmul(
            softmax_output + i * S * S,
            V + i * S * D,
            final_output + i * S * D,
            S, D, S
        );
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&t_v_matmul, start, stop);
    std::cout << "Softmax × V time: " << t_v_matmul << " ms" << std::endl;

    float total_time = t_transpose + t_qk_matmul + t_softmax + t_v_matmul;
    std::cout << "Total: " << total_time << " ms" << std::endl;

    hipFree(Q);
    hipFree(K);
    hipFree(V);
    hipFree(K_T);
    hipFree(attn_scores);
    hipFree(softmax_output);
    hipFree(final_output);

    return 0;
}
