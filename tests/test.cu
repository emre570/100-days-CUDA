#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32  // Block boyutunu değiştirebilirsin

// CUDA Kernel
__global__ void matMulKernel(float *A, float *B, float *C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void runTest(int M, int N, int K) {
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    std::cout << "Device Name: " << prop.name << std::endl;

    size_t sizeA = M * K * sizeof(float);
    size_t sizeB = K * N * sizeof(float);
    size_t sizeC = M * N * sizeof(float);

    float *h_A = (float *)malloc(sizeA);
    float *h_B = (float *)malloc(sizeB);
    float *h_C = (float *)malloc(sizeC);

    for (int i = 0; i < M * K; i++) h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    for (int i = 0; i < K * N; i++) h_B[i] = static_cast<float>(rand()) / RAND_MAX;

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // CUDA kernel süresini ölçmek için event'ler
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    hipEventRecord(startEvent);
    matMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);
    hipEventRecord(stopEvent);
    hipEventSynchronize(stopEvent);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startEvent, stopEvent);

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    std::cout << "CUDA Execution Time: " << milliseconds << " ms" << std::endl;

    // TFLOPS hesaplama
    long long int FLOP = 2LL * M * N * K;
    double TFLOPS = (FLOP / (milliseconds / 1000.0)) / 1e12;

    std::cout << "Achieved Performance: " << TFLOPS << " TFLOPS" << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
}

int main() {
    int M = 1024, N = 1024, K = 1024;
    runTest(M, N, K);
    return 0;
}
